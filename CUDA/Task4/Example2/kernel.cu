#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <locale.h>
const int N = 1024 * 33;
const int BLOCK_SIZE = 256;
const int blocksPerGrid = N / BLOCK_SIZE;

__global__ void mult2(int* a, int* b, int* c)
{
	__shared__ float cache[BLOCK_SIZE];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float temp = 0;
	while (i < N)
	{
		temp += a[i] * b[i];
		i += blockDim.x * gridDim.x;
	}
	cache[tid] = temp;
	__syncthreads();
	for (int s = 1; s < blockDim.x; s <<= 1)
	{
		int index = 2 * s * tid;
		if (index < blockDim.x) 
			cache[index] += cache[index + s];
		__syncthreads();
	}
	if (tid == 0)
		c[blockIdx.x] = cache[0];
}

/*__global__ void reduce2(int* inData, int* outData)
{
	__shared__ int data[BLOCK_SIZE];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	data[tid] = inData[i]; //load into shared memory
	__syncthreads();
	for (int s = 1; s < blockDim.x; s <<= 1)
	{
		int index = 2 * s * tid;
		if (index < blockDim.x) data[index] += data[index + s];
		__syncthreads();
	}
	if (tid == 0) //write result of block reduction
		outData[blockIdx.x] = data[0];
}*/

int main(void)
{
	setlocale(LC_ALL, "RUS");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int* a, * b, * partial_c, c;
	int* dev_a, * dev_b, * dev_partial_c;
	a = (int*)malloc(N * sizeof(int));
	b = (int*)malloc(N * sizeof(int));
	partial_c = (int*)malloc(blocksPerGrid * sizeof(int));
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(int));
	for (int i = 0; i < N; i++)
	{
		a[i] = 1;
		b[i] = 2;
	}
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	mult2 <<<blocksPerGrid, BLOCK_SIZE>>>(dev_a, dev_b, dev_partial_c);
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
		c += partial_c[i];
	printf("Скалярное произведение, вычисленное на GPU, равно %d\n", c);
	printf("Время выполнения: %.6f миллисекунд\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
	free(a);
	free(b);
	free(partial_c);
	return 0;
}
