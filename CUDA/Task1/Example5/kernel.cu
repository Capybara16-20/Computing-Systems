﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#define N 10

__global__ void kernel() 
{
	//обработать данные, находящиеся по этому индексу
	int tid = threadIdx.x;
	printf("Thread number %d\n", tid);
}

int main(void)
{
	int* dev_a;
	//выделение памятина GPU
	hipMalloc((void**)&dev_a, sizeof(int));
	kernel<<<1, N>>>(); //вызов ядра
	// освобождение памяти на устройстве GPU
	hipFree(dev_a);
	return 0;
}