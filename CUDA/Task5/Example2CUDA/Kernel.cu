#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "hip/device_functions.h"

__global__ void kernel(unsigned* src, unsigned* red, unsigned* green,
	unsigned* blue, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x < width) && (y < height))
	{
		unsigned pixel = src[y * width + x];
		unsigned gray = ((pixel & 0x00FF0000 >> 0x10) +
			(pixel & 0x0000FF00 >> 0x8) + (pixel & 0x000000FF)) / 3;
		unsigned graypixel = 0xFF000000 | (gray << 0x10) | (gray << 0x8) | gray;
		red[y * width + x] = graypixel | 0x00FF0000;
		green[y * width + x] = graypixel | 0x0000FF00;
		blue[y * width + x] = graypixel | 0x000000FF;
	}
}

int calc(unsigned* srcImage, unsigned* redImage, unsigned* greenImage,
	unsigned* blueImage, int width, int height)
{
	unsigned* dev_srcImage;
	unsigned* dev_redImage;
	unsigned* dev_greenImage;
	unsigned* dev_blueImage;
	size_t size = sizeof(unsigned) * width * height;
	if ((hipMalloc((void**)&dev_srcImage, size) != hipError_t::hipSuccess) ||
		(hipMalloc((void**)&dev_redImage, size) != hipError_t::hipSuccess) ||
		(hipMalloc((void**)&dev_greenImage, size) != hipError_t::hipSuccess) ||
		(hipMalloc((void**)&dev_blueImage, size) != hipError_t::hipSuccess))
		return -1;
	if ((hipMemcpy(dev_srcImage, srcImage, size,
		hipMemcpyHostToDevice) != hipError_t::hipSuccess) ||
		(hipMemcpy(dev_redImage, redImage, size,
			hipMemcpyHostToDevice) != hipError_t::hipSuccess) ||
		(hipMemcpy(dev_greenImage, greenImage, size,
			hipMemcpyHostToDevice) != hipError_t::hipSuccess) ||
		(hipMemcpy(dev_blueImage, blueImage, size,
			hipMemcpyHostToDevice) != hipError_t::hipSuccess))
		return -1;
	// ������������ ���������� ����� �� ���� ����� ����������
	dim3 threads(128, 128);
	dim3 blocks((width + threads.x - 1) / threads.x,
		(height + threads.y - 1) / threads.y);
	kernel << <threads, blocks >> > (dev_srcImage, dev_redImage,
		dev_greenImage, dev_blueImage, width, height);
	hipMemcpy(srcImage, dev_srcImage, size, hipMemcpyDeviceToHost);
	hipMemcpy(redImage, dev_redImage, size, hipMemcpyDeviceToHost);
	hipMemcpy(greenImage, dev_greenImage, size, hipMemcpyDeviceToHost);
	hipMemcpy(blueImage, dev_blueImage, size, hipMemcpyDeviceToHost);
	hipFree(dev_srcImage);
	hipFree(dev_redImage);
	hipFree(dev_greenImage);
	hipFree(dev_blueImage);
	return 0;
}

