﻿#include "hip/hip_runtime.h"
#include ""
#include <locale.h>
#include <stdio.h>
int main()
{
	hipDeviceProp_t prop;
	setlocale(LC_ALL, "RUS");
	int count;
	hipGetDeviceCount(&count);
	printf("Количество устройств: %d\n\n", count);
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);
		printf("- Общая информация об устройстве %d -\n", i);
		printf("	Имя: %s\n", prop.name);
		printf("	Вычислительные возможности: %d.%d\n", prop.major, prop.minor);
		printf("	Тактовая частота: %d\n", prop.clockRate);
		printf("	Перекрытие копирования: ");
		if (prop.deviceOverlap)
			printf("разрешено\n");
		else printf("запрещено\n");
		printf("	Тайм-аут выполнения ядра: ");
		if (prop.kernelExecTimeoutEnabled)
			printf("включен\n\n");
		else
			printf("выключен\n\n");
		printf("- Информация о памяти для устройства %d -\n", i);
		printf("	Всего глобальной памяти: %zu\n", prop.totalGlobalMem);
		printf("	Всего константной памяти: %zu\n", prop.totalConstMem);
		printf("	Максимальный шаг: %zu\n", prop.memPitch);
		printf("	Выравнивание текстур: %zu\n\n", prop.textureAlignment);
		printf("- Инфо о мультипроцессорах для устройства %d -\n", i);
		printf("	Кол-во мультипроцессоров: %d\n", prop.multiProcessorCount);
		printf("	Разделяемая память на один МП: %zu\n", prop.sharedMemPerBlock);
		printf("	Количество регистров на один МП: %d\n", prop.regsPerBlock);
		printf("	Количество нитей в варпе: %d\n", prop.warpSize);
		printf("	Макс.кол-во нитей в блоке: %d\n", prop.maxThreadsPerBlock);
		printf("	Макс.кол-во нитей по измерениям: (%d, %d,% d)\n", prop.maxThreadsDim[0],
			prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("	Максимальные размеры сетки: (%d, %d, %d)\n", prop.maxGridSize[0],
			prop.maxGridSize[1], prop.maxGridSize[2]);
	}
}

