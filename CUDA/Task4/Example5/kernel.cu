﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <locale.h>
const int N = 1024 * 33;
const int BLOCK_SIZE = 256;
const int blocksPerGrid = N / BLOCK_SIZE;

__global__ void mult5(int* a, int* b, int* c)
{
	__shared__ float cache[BLOCK_SIZE];
	int tid = threadIdx.x;
	int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	//Записать сумму первых двух элементов в разделяемую память
	float temp = 0;
	while (i < N)
	{
		temp += a[i] * b[i];
		temp += a[i + blockDim.x] * b[i + blockDim.x];
		i += blockDim.x * gridDim.x;
	}
	cache[tid] = temp;
	__syncthreads();
	for (int s = blockDim.x / 2; s > 32; s >>= 1)
	{
		if (tid < s)
			cache[tid] += cache[tid + s];
		__syncthreads();
	}
	if (tid < 32)
	{
		cache[tid] += cache[tid + 32];
		cache[tid] += cache[tid + 16];
		cache[tid] += cache[tid + 8];
		cache[tid] += cache[tid + 4];
		cache[tid] += cache[tid + 2];
		cache[tid] += cache[tid + 1];
	}
	if (tid == 0)
		c[blockIdx.x] = cache[0];
}

/*__global__ void reduce5(int* inData, int* outData)
{
	__shared__ int data[BLOCK_SIZE];
	int tid = threadIdx.x;
	int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	//Записать сумму первых двух элементов в разделяемую память
	data[tid] = inData[i] + inData[i + blockDim.x];
	__syncthreads();
	for (int s = blockDim.x / 2; s > 32; s >>= 1)
	{
		if (tid < s)
			data[tid] += data[tid + s];
		__syncthreads();
	}
	if (tid < 32)
	{
		data[tid] += data[tid + 32];
		data[tid] += data[tid + 16];
		data[tid] += data[tid + 8];
		data[tid] += data[tid + 4];
		data[tid] += data[tid + 2];
		data[tid] += data[tid + 1];
	}
	if (tid == 0)
		outData[blockIdx.x] = data[0];
}*/

int main(void)
{
	setlocale(LC_ALL, "RUS");
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	int* a, * b, * partial_c, c;
	int* dev_a, * dev_b, * dev_partial_c;
	a = (int*)malloc(N * sizeof(int));
	b = (int*)malloc(N * sizeof(int));
	partial_c = (int*)malloc(blocksPerGrid * sizeof(int));
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(int));
	for (int i = 0; i < N; i++)
	{
		a[i] = 1;
		b[i] = 2;
	}
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	mult5 <<<blocksPerGrid, BLOCK_SIZE>>>(dev_a, dev_b, dev_partial_c);
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
		c += partial_c[i];
	printf("Скалярное произведение, вычисленное на GPU, равно %d\n", c);
	printf("Время выполнения: %.6f миллисекунд\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
	free(a);
	free(b);
	free(partial_c);
	return 0;
}
