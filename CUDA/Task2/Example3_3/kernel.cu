﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include<locale.h>
#define N 16

__global__ void getAverage(float* matrix, float* average)
{
	__shared__ float cache[N];
	int tid_block = blockIdx.x;
	int tid_thread = threadIdx.x;
	cache[tid_thread] = matrix[tid_block * N + tid_thread] / N;
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (tid_thread < i)
			cache[tid_thread] += cache[tid_thread + i];
		__syncthreads();
		i /= 2;
	}
	if (tid_thread == 0)
		average[blockIdx.x] = cache[0];
}

int main(void)
{
	setlocale(LC_ALL, "Rus");
	float matrix[N * N];
	float* dev_matrix;
	float average[N];
	float* dev_average;
	hipMalloc((void**)&dev_matrix, N * N * sizeof(float));
	hipMalloc((void**)&dev_average, N * sizeof(float));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			matrix[i * N + j] = i + j;
	hipMemcpy(dev_matrix, matrix, N * N * sizeof(float), hipMemcpyHostToDevice);
	printf("Матрица:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%.0f\t", matrix[i * N + j]);
		printf("\n");
	}
	getAverage <<<N, N>>>(dev_matrix, dev_average);
	hipMemcpy(average, dev_average, N * sizeof(float), hipMemcpyDeviceToHost);
	printf("\n");
	for (int i = 0; i < N; i++)
		printf("Среднее арефмитическое %d-й строки: %.1f\n", i + 1, average[i]);
	hipFree(dev_matrix);
	hipFree(dev_average);
	return 0;
}

