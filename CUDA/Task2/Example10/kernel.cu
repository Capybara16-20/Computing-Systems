﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <locale.h>
#define N 16

const int threadsPerBlock = 4;
const int blocksCount = 4;
__global__ void kernel(float* a, float* b, float* c)
{
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * threadsPerBlock;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while (tid < N)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main()
{
    setlocale(LC_ALL, "Rus");
    float a[N], b[N], c[N];
    float* dev_a, * dev_b, * dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, N * sizeof(float));
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i + 1;
    }
    for (int i = 0; i < N; i++)
        printf("%f * %f\n", a[i], b[i]);
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    kernel <<<threadsPerBlock, blocksCount>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);
    float mult = 0;
    for (int i = 0; i < blocksCount; i++)
        mult += c[i];
    printf("\nСкалярное произведение: %f\n", mult);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}


