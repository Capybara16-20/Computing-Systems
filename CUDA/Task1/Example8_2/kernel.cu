﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include<locale.h>
#define N 15

__global__ void row_sum(int* matrix, int* sum)
{
	__shared__ float cache[hipDeviceAttributeMaxThreadsPerBlock];
	int tid_block = blockIdx.x;
	int tid_thread = threadIdx.x;
	cache[tid_thread] = matrix[tid_block * N + tid_thread];
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (tid_thread < i)
			cache[tid_thread] += cache[tid_thread + i];
		__syncthreads();
		i /= 2;
	}
	if (tid_thread == 0)
		sum[blockIdx.x] = cache[0];
}

int main(void)
{
	setlocale(LC_ALL, "Rus");
	int matrix[N * N];
	int* dev_matrix;
	int sum[N];
	int* dev_sum;
	hipMalloc((void**)&dev_matrix, N * N * sizeof(int));
	hipMalloc((void**)&dev_sum, N * sizeof(int));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			matrix[i * N + j] = i + j;
	hipMemcpy(dev_matrix, matrix, N * N * sizeof(int), hipMemcpyHostToDevice);
	printf("Матрица:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%d\t", matrix[i * N + j]);
		printf("\n");
	}
	row_sum <<<N, N>>>(dev_matrix, dev_sum);
	hipMemcpy(sum, dev_sum, N * sizeof(int), hipMemcpyDeviceToHost);
	printf("\n");
	for (int i = 0; i < N; i++)
		printf("Сумма %d-й строки: %d\n", i + 1, sum[i]);
	hipFree(dev_matrix);
	hipFree(dev_sum);
	return 0;
}

