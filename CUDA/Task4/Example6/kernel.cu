#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <locale.h>
const int N = 1024 * 33;
const int BLOCK_SIZE = 256;
const int blocksPerGrid = N / BLOCK_SIZE;

__global__ void kernel(int* a, int* b, int* c)
{
	__shared__ float cache[BLOCK_SIZE];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float temp = 0;
	while (i < N)
	{
		temp += a[i] * b[i];
		i += blockDim.x * gridDim.x;
	}
	cache[tid] = temp;
	__syncthreads();
	for (int s = 1; s < blockDim.x; s *= 2)
	{
		//проверuть, участвует лu нить на данном шаге
		if (tid % (2 * s) == 0)
			cache[tid] += cache[tid + s];
		__syncthreads();
	}
	if (tid == 0)
		c[blockIdx.x] = cache[0];
}

int reduce(int* data, int N)
{
	int* sums = NULL;
	int numBlocks = N / BLOCK_SIZE;
	int res = 0;
	//Выделumь памяmь под массив сумм блоков.
	hipMalloc((void**)&sums, numBlocks * sizeof(int));
	//Провесmu поблочную редукцuю, запuсав суммы
	//для каждого блока в массив sums
	kernel <<<dim3(numBlocks), dim3(BLOCK_SIZE)>>>(data, sums);
	//Теперь редуцируем массив сумм для блоков
	if (numBlocks > BLOCK_SIZE)
	{
		res = reduce(sums, numBlocks);
	}
	else 
	{
		//Еслu значенuе мало, то просуммируем явно
		int* sumsHost = new int[numBlocks];
		hipMemcpy(sumsHost, sums, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
		for (int i = 0; i < numBlocks; i++)
			res += sumsHost[i];
		delete[] sumsHost;
	}
	hipFree(sums);
	return res;
}

int main(void) 
{
	setlocale(LC_ALL, "RUS");
	
	return 0;
}