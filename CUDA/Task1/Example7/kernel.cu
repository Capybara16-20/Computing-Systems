#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#define CUDA_FLOAT float
#define GRID_SIZE 256
#define BLOCK_SIZE 256

__global__ void pi_kern(CUDA_FLOAT* res)
{
	int n = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	// Начало отрезка интегрирования
	CUDA_FLOAT x0 = n * 1.f / (BLOCK_SIZE * GRID_SIZE);
	CUDA_FLOAT y0 = sqrtf(1 - x0 * x0);
	CUDA_FLOAT dx = 1.f / (1.f * BLOCK_SIZE * GRID_SIZE); // Шаги интегрирования
	// Значение интеграла по отрезку, данному текущему треду
	CUDA_FLOAT s = 0;
	CUDA_FLOAT x1, y1;
	x1 = x0 + dx;
	y1 = sqrtf(1 - x1 * x1);
	s = (y0 + y1) * dx / 2.f; // Площадь трапеции
	res[n] = s; // Запись результата в глобальную память
}

int main() 
{
	CUDA_FLOAT pi = 0;
	CUDA_FLOAT* res_d; // Результаты на устройстве
	CUDA_FLOAT res[GRID_SIZE * BLOCK_SIZE]; // Результаты в host памяти
	// Выделение памяти на CPU
	hipMalloc((void**)&res_d, sizeof(CUDA_FLOAT) * GRID_SIZE * BLOCK_SIZE);
	// Размеры грида и блока на GPU
	dim3 grid(GRID_SIZE);
	dim3 block(BLOCK_SIZE);
	pi_kern<<<grid, block>>>(res_d); // Запуск ядра
	hipDeviceSynchronize(); // Ожидаем завершения работы ядра
	hipMemcpy(&res, res_d, sizeof(CUDA_FLOAT) * GRID_SIZE * BLOCK_SIZE, 
		hipMemcpyDeviceToHost); // Копируем результаты на хост
	hipFree(res_d); // Освобождаем память на GPU
	for (int i = 0; i < GRID_SIZE * BLOCK_SIZE; i++)
		pi += res[i];
	pi *= 4;
	printf("PI = %f\n", pi);
	return 0;
}


