﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <locale.h>
#define BS 16//Размер блока
#define BC 4//Количество блоков

__global__ void kern(float* data)
{
    //Создается массив float в разделяемой памяти
    __shared__ float a[BS];
    int idx = blockIdx.x * BS + threadIdx.x;
    //Копируем из глобальной памяти в разделяемую
    a[threadIdx.x] = data[idx];
    //Перед использованием надо быть уверенным,
    //что все данные скопированы
    __syncthreads(); //Синхронизируем
    data[idx] = a[threadIdx.x] + a[(threadIdx.x + 1) % BS]; //Используем
}

int main()
{
    setlocale(LC_ALL, "Rus");
    float data[BC * BS];
    float* dev_data;
    hipMalloc((void**)&dev_data, BC * BS * sizeof(float));
    for (int i = 0; i < BC * BS; i++)
        data[i] = i;
    hipMemcpy(dev_data, data, BC * BS * sizeof(float), hipMemcpyHostToDevice);
    kern <<<BC, BS>>>(dev_data);
    hipMemcpy(data, dev_data, BC * BS * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < BC * BS; i++)
        printf("%f\n", data[i]);
    hipFree(dev_data);
    return 0;
}
