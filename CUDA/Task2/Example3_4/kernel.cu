#include "hip/hip_runtime.h"
﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<locale.h>
#define N 16

__global__ void squaresSum(float* matrix, float* row_sum, float* col_sum)
{
	__shared__ float row_cache[N], col_cache[N];
	int tid_row = threadIdx.x + blockIdx.x * N;
	int tid_col = blockIdx.x + threadIdx.x * N;
	row_cache[threadIdx.x] = matrix[tid_row] * matrix[tid_row];
	col_cache[threadIdx.x] = matrix[tid_col] * matrix[tid_col];
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (threadIdx.x < i)
		{
			row_cache[threadIdx.x] += row_cache[threadIdx.x + i];
			col_cache[threadIdx.x] += col_cache[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}
	if (threadIdx.x == 0)
	{
		row_sum[blockIdx.x] = row_cache[0];
		col_sum[blockIdx.x] = col_cache[0];
	}
}

int main(void)
{
	setlocale(LC_ALL, "Rus");
	float matrix[N * N];
	float* dev_matrix;
	float row_sum[N];
	float* dev_row_sum;
	float col_sum[N];
	float* dev_col_sum;
	hipMalloc((void**)&dev_matrix, N * N * sizeof(float));
	hipMalloc((void**)&dev_row_sum, N * sizeof(float));
	hipMalloc((void**)&dev_col_sum, N * sizeof(float));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			matrix[i * N + j] = i + j;
	hipMemcpy(dev_matrix, matrix, N * N * sizeof(float), hipMemcpyHostToDevice);
	printf("Матрица:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%.0f\t", matrix[i * N + j]);
		printf("\n");
	}
	squaresSum << <N, N >> > (dev_matrix, dev_row_sum, dev_col_sum);
	hipMemcpy(row_sum, dev_row_sum, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(col_sum, dev_col_sum, N * sizeof(float), hipMemcpyDeviceToHost);
	printf("\n");
	for (int i = 0; i < N; i++)
		printf("Сумма квадратов %d-й строки: %.0f  \tСумма квадратов %d-го столбца: %.0f\n",
			i + 1, row_sum[i], i + 1, col_sum[i]);
	hipFree(dev_matrix);
	hipFree(dev_row_sum);
	return 0;
}
