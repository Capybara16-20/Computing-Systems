#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
const int maxThreads = 512;

__device__ double GetEuclideanDistance(unsigned color1, unsigned color2)
{
	unsigned red1 = color1 & 0x00FF0000 >> 0x10;
	unsigned green1 = color1 & 0x0000FF00 >> 0x8;
	unsigned blue1 = color1 & 0x000000FF;
	unsigned red2 = color2 & 0x00FF0000 >> 0x10;
	unsigned green2 = color2 & 0x0000FF00 >> 0x8;
	unsigned blue2 = color2 & 0x000000FF;

	return (red1 - red2) * (red1 - red2) + (green1 - green2) * (green1 - green2) + (blue1 - blue2) * (blue1 - blue2);
}

__global__ void calculateClusters_kernel(unsigned* src, int width, int height, 
	int clustersCount, int* centers_x, int* centers_y, int* pointsClusters, int* changesCount)
{
	//��� ������ �����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < width * height)
	{
		unsigned current_color = src[i];
		int nearestCluster_id = 0;
		unsigned cluster_color = src[centers_y[0] * width + centers_x[0]];
		double minDistance = GetEuclideanDistance(current_color, cluster_color);

		//���� ��������� �������
		for (int k = 1; k < clustersCount; k++)
		{
			cluster_color = src[centers_y[k] * width + centers_x[k]];
			double cl_distance = GetEuclideanDistance(current_color, cluster_color);
			if (cl_distance < minDistance)
			{
				minDistance = cl_distance;
				nearestCluster_id = k;
			}
		}

		//�������� ����� � ��������� �������
		int currentCluster = pointsClusters[i];
		if (nearestCluster_id != currentCluster)
		{
			pointsClusters[i] = nearestCluster_id;
			*changesCount++;
		}
	}
}

__global__ void initializePointsClusters_kernel(int* pointsClusters, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height)
	{
		pointsClusters[y * width + x] = -1;
	}
}

__global__ void initializeColors_kernel(unsigned* colors, int clustersCount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < clustersCount)
	{
		colors[i] = 0;
	}
}

__global__ void calculateColors_kernel(unsigned* src, int width, int height, 
	int* pointsClusters, unsigned* colors)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < width * height)
	{
		unsigned pixel = src[i];
		int cluster = pointsClusters[i];

		//colors[cluster] |= (pixel & 0x00FF0000 >> 0x10 + pixel & 0x0000FF00 >> 0x8 + pixel & 0x000000FF);
		colors[cluster] |= pixel;
	}
}

__global__ void fillColors_kernel(unsigned* colors, int clustersCount)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < clustersCount)
	{
		//colors[i] = 0xFF000000 | (colors[i] << 0x10) | (colors[i] << 0x8) | colors[i];
		colors[i] = 0xFF000000 | colors[i];
	}
}

__global__ void paint_kernel(unsigned* src, int width, int height, unsigned* colors, int* pointsClusters)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < width * height)
	{
		src[i] = colors[pointsClusters[i]];
	}
}

int calc(unsigned* srcImage, int width, int height, int clustersCount)
{
	srand(time(NULL));

	//�������������� �������� �����, ����� � ���������� ����� �� ���������
#pragma region
	int* pointsClusters = new int[width * height];
	int* dev_pointsClusters;
	hipMalloc((void**)&dev_pointsClusters, sizeof(int) * width * height);
	hipMemcpy(dev_pointsClusters, pointsClusters, sizeof(int) * width * height, hipMemcpyHostToDevice);

	dim3 threads;
	dim3 blocks;
	if (width * height < maxThreads)
	{
		threads = dim3(width, height);
		blocks = dim3();
	}
	else
	{
		threads = dim3(maxThreads, maxThreads);
		blocks = dim3((width + maxThreads - 1) / maxThreads, (height + maxThreads - 1) / maxThreads);
	}

	initializePointsClusters_kernel <<<blocks, threads>>> (dev_pointsClusters, width, height);
	hipMemcpy(pointsClusters, dev_pointsClusters, sizeof(int) * width * height, hipMemcpyDeviceToHost);

	hipFree(dev_pointsClusters);
#pragma endregion

	//�������� ��������� ��������
#pragma region
	int* centers_x = new int[clustersCount];
	int* centers_y = new int[clustersCount];
	for (int i = 0; i < clustersCount; i++)
	{
		centers_x[i] = rand() % width;
		centers_y[i] = rand() % height;
	}

	int blockSize;
	int gridSize;
	if (width * height < maxThreads)
	{
		blockSize = width * height;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (width * height) / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	unsigned* dev_srcImage;
	hipMalloc((void**)&dev_srcImage, sizeof(unsigned) * width * height);
	hipMemcpy(dev_srcImage, srcImage, sizeof(unsigned) * width * height, hipMemcpyHostToDevice);
	bool complited = false;
	while (!complited) //���� �� ����������� ���������
	{
		int changesCount = 0;
		int* dev_changesCount;
		hipMalloc((void**)&dev_changesCount, sizeof(int));
		hipMemcpy(dev_changesCount, &changesCount, sizeof(int), hipMemcpyHostToDevice);
		
		hipMalloc((void**)&dev_pointsClusters, sizeof(int) * width * height);
		hipMemcpy(dev_pointsClusters, pointsClusters, sizeof(int) * width * height, hipMemcpyHostToDevice);

		int* dev_centers_x;
		hipMalloc((void**)&dev_centers_x, sizeof(int) * clustersCount);
		hipMemcpy(dev_centers_x, centers_x, sizeof(int) * clustersCount, hipMemcpyHostToDevice);
		int* dev_centers_y;
		hipMalloc((void**)&dev_centers_y, sizeof(int) * clustersCount);
		hipMemcpy(dev_centers_y, centers_y, sizeof(int) * clustersCount, hipMemcpyHostToDevice);

		calculateClusters_kernel <<<blocks, threads>>> (dev_srcImage, width, height, clustersCount, 
			dev_centers_x, dev_centers_y, dev_pointsClusters, dev_changesCount);
		hipMemcpy(centers_x, dev_centers_x, sizeof(int) * clustersCount, hipMemcpyDeviceToHost);
		hipMemcpy(centers_y, dev_centers_y, sizeof(int) * clustersCount, hipMemcpyDeviceToHost);
		hipMemcpy(pointsClusters, dev_pointsClusters, sizeof(int) * width * height, hipMemcpyDeviceToHost);
		hipMemcpy(&changesCount, dev_changesCount, sizeof(int), hipMemcpyDeviceToHost);

		hipFree(dev_centers_x);
		hipFree(dev_centers_y);
		hipFree(dev_pointsClusters);
		hipFree(dev_changesCount);

		if (changesCount == 0) //���� ��������� �� ����
			complited = true; //�� ����� ������������ �� ���������
	}
	hipFree(dev_srcImage);
#pragma endregion

	//��������� ����� ������� ��������
#pragma region
	//������������� ������
	if (clustersCount < maxThreads)
	{
		blockSize = clustersCount;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = clustersCount / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	unsigned* colors = new unsigned[clustersCount];
	unsigned* dev_colors;
	hipMalloc((void**)&dev_colors, sizeof(unsigned) * clustersCount);

	initializeColors_kernel <<<blocks, threads>>> (dev_colors, clustersCount);
	hipMemcpy(colors, dev_colors, sizeof(unsigned) * clustersCount, hipMemcpyDeviceToHost);
	
	//���������� ������
	if (width * height < maxThreads)
	{
		blockSize = width * height;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (width * height) / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	hipMalloc((void**)&dev_colors, sizeof(unsigned) * clustersCount);

	hipMalloc((void**)&dev_pointsClusters, sizeof(int) * width * height);
	hipMemcpy(dev_pointsClusters, pointsClusters, sizeof(int) * width * height, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_srcImage, sizeof(unsigned) * width * height);
	hipMemcpy(dev_srcImage, srcImage, sizeof(unsigned) * width * height, hipMemcpyHostToDevice);

	calculateColors_kernel <<<blocks, threads>>> (dev_srcImage, width, height,
		dev_pointsClusters, dev_colors);
	hipMemcpy(colors, dev_colors, sizeof(unsigned) * clustersCount, hipMemcpyDeviceToHost);

	if (clustersCount < maxThreads)
	{
		blockSize = clustersCount;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = clustersCount / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	hipMalloc((void**)&dev_colors, sizeof(unsigned) * clustersCount);
	hipMemcpy(dev_colors, colors, sizeof(unsigned)* clustersCount, hipMemcpyHostToDevice);

	fillColors_kernel <<<blocks, threads>>> (dev_colors, clustersCount);
	hipMemcpy(colors, dev_colors, sizeof(unsigned)* clustersCount, hipMemcpyDeviceToHost);

	hipFree(dev_srcImage);
	hipFree(dev_pointsClusters);
	hipFree(dev_colors);
#pragma endregion

	//���������� ��������� �����������
#pragma region
	if (width * height < maxThreads)
	{
		blockSize = width * height;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (width * height) / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	hipMalloc((void**)&dev_srcImage, sizeof(unsigned)* width* height);
	hipMemcpy(dev_srcImage, srcImage, sizeof(unsigned)* width* height, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_colors, sizeof(unsigned)* clustersCount);
	hipMemcpy(dev_colors, colors, sizeof(unsigned)* clustersCount, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_pointsClusters, sizeof(int)* width* height);
	hipMemcpy(dev_pointsClusters, pointsClusters, sizeof(int)* width* height, hipMemcpyHostToDevice);

	paint_kernel <<<blocks, threads>>> (dev_srcImage, width, height, dev_colors, dev_pointsClusters);
	hipMemcpy(srcImage, dev_srcImage, sizeof(unsigned)* width* height, hipMemcpyDeviceToHost);

	hipFree(dev_srcImage);
	hipFree(dev_colors);
	hipFree(dev_pointsClusters);
#pragma endregion

	return 0;
}


