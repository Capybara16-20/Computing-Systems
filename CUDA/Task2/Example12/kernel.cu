﻿
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <locale.h>
#include <stdlib.h>
//функция определения наименьшего значения
#define imin(a, b) (a < b ? a : b)

const int N = 33 * 1024; //количество элементов в векторе
const int threadsPerBlock = 256; //количество нитей в блоке
//количество блоков в гриде
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);
__global__ void kernel(float* a, float* b, float* c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * threadsPerBlock;
	int cacheIndex = threadIdx.x;
	float temp = 0;
	while (tid < N)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}
	cache[cacheIndex] = temp;
	__syncthreads();
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main(void)
{
	setlocale(LC_ALL, "RUS");
	float* a, * b, c, * partial_c;
	float* dev_a, * dev_b, * dev_partial_c;
	//выделение памяти на CPU
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));
	//выделение памяти на GPU
	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float));
	//заполнение массивов данными на хосте
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * 2;
	}
	//копирование массивов 'a' и 'b' на GPU
	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
	kernel <<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
	//копирование массива 'c' с GPU назад на CPU
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
	//окончание вычислений на CPU
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
		c += partial_c[i];
	printf("Скалярное произведение векторов, вычисленное на GPU, равно % .6f\n", c);
	//освобождение памяти на GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
	//освобождение памяти на CPU
	free(a);
	free(b);
	free(partial_c);
	return 0;
}

