#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int maxThreads = 128;
const int maxCount = 1024;

__global__ void getGrayImage_kernel(unsigned* src, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x < width) && (y < height))
	{
		unsigned pixel = src[y * width + x];
		unsigned gray = ((pixel & 0x00FF0000 >> 0x10) +
			(pixel & 0x0000FF00 >> 0x8) + (pixel & 0x000000FF)) / 3;
		src[y * width + x] = 0xFF000000 | (gray << 0x10) | (gray << 0x8) | gray;
	}
}

__global__ void getRandomNumbers_kernel(unsigned int seed, hiprandState_t* states, 
	int iterations, int pointsCount, int coordsCount, int maxX, int maxY, int* coords)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < iterations * pointsCount * coordsCount)
	{
		hiprand_init(seed, i, 0, &states[i]);
		if (i % 2 == 0)
			coords[i] = hiprand(&states[i]) % maxX;
		else
			coords[i] = hiprand(&states[i]) % maxY;
	}
}

__global__ void getCoords(int iterations, int pointsCount, int coordsCount, int* coords, int* xCoords, int* yCoords)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = tid * coordsCount;
	xCoords[tid] = coords[i];
	yCoords[tid] = coords[i + 1];
}

__global__ void getFunctionParameters(int x, int y, unsigned* main, int mainWidth, 
	int mainHeight, unsigned* sub, int subWidth, int subHeight, 
	long double* numerators, long double* subMultipliers, long double* mainMultipliers)
{
	/*int subX = blockIdx.x * blockDim.x + threadIdx.x;
	int subY = blockIdx.y * blockDim.y + threadIdx.y;

	unsigned subPixel = sub[subY * subWidth + subX];
	float subBrightness = 0.299 * ((subPixel & 0x00ff0000) >> 16)
		+ 0.587 * ((subPixel & 0x0000ff00) >> 8) + 0.114 * (subPixel & 0x000000ff);

	unsigned mainPixel = main[(y + subY) * mainWidth + x + subX];
	float mainBrightness = 0.299 * ((mainPixel & 0x00ff0000) >> 16)
		+ 0.587 * ((mainPixel & 0x0000ff00) >> 8) + 0.114 * (mainPixel & 0x000000ff);

	numerators[subY * subWidth + subX] = subBrightness * mainBrightness;
	subMultipliers[subY * subWidth + subX] = subBrightness * subBrightness;
	mainMultipliers[subY * subWidth + subX] = mainBrightness * mainBrightness;*/

	__shared__ long double temp_numerators[maxThreads];
	__shared__ long double temp_subMultipliers[maxThreads];
	__shared__ long double temp_mainMultipliers[maxThreads];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	long double temp_numerator = 0;
	long double temp_subMultiplier = 0;
	long double temp_mainMultiplier = 0;
	while (i < subWidth * subHeight)
	{
		unsigned subPixel = sub[blockIdx.x * blockDim.x + threadIdx.x];
		long double subBrightness = 0.299 * ((subPixel & 0x00ff0000) >> 16)
			+ 0.587 * ((subPixel & 0x0000ff00) >> 8) + 0.114 * (subPixel & 0x000000ff);

		unsigned mainPixel = main[(blockIdx.x + y) * blockDim.x + threadIdx.x + x];
		long double mainBrightness = 0.299 * ((mainPixel & 0x00ff0000) >> 16)
			+ 0.587 * ((mainPixel & 0x0000ff00) >> 8) + 0.114 * (mainPixel & 0x000000ff);

		temp_numerator += subBrightness * mainBrightness;
		temp_subMultiplier += subBrightness * subBrightness;
		temp_mainMultiplier += mainBrightness * mainBrightness;

		i += blockDim.x * gridDim.x;
	}
	temp_numerators[tid] = temp_numerator;
	temp_subMultipliers[tid] = temp_subMultiplier;
	temp_mainMultipliers[tid] = temp_mainMultiplier;
	__syncthreads();
	for (int s = 1; s < blockDim.x; s *= 2)
	{
		//������u��, ��������� �u ���� �� ������ ����
		if (tid % (2 * s) == 0)
		{
			temp_numerators[tid] += temp_numerators[tid + s];
			temp_subMultipliers[tid] += temp_subMultipliers[tid + s];
			temp_mainMultipliers[tid] += temp_mainMultipliers[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		numerators[blockIdx.x] = temp_numerators[0];
		subMultipliers[blockIdx.x] = temp_subMultipliers[0];
		mainMultipliers[blockIdx.x] = temp_mainMultipliers[0];
	}
}

int getGrayImage(unsigned* srcImage, int width, int height)
{
	unsigned* dev_srcImage;
	size_t size = sizeof(unsigned) * width * height;
	if (hipMalloc((void**)&dev_srcImage, size) != hipError_t::hipSuccess)
		return -1;
	if (hipMemcpy(dev_srcImage, srcImage, size,
		hipMemcpyHostToDevice) != hipError_t::hipSuccess)
		return -1;

	dim3 threads;
	dim3 blocks;
	if (width * height < maxThreads)
	{
		threads = dim3(width, height);
		blocks = dim3();
	}
	else
	{
		threads = dim3(maxThreads, maxThreads);
		blocks = dim3((width + maxThreads - 1) / maxThreads, (height + maxThreads - 1) / maxThreads);
	}

	getGrayImage_kernel <<<blocks, threads>>> (dev_srcImage, width, height);

	if (hipMemcpy(srcImage, dev_srcImage, size, 
		hipMemcpyDeviceToHost) != hipError_t::hipSuccess)
		return -1;

	hipFree(dev_srcImage);
	return 0;
}

int findImage(unsigned* mainImage, int mainWidth, int mainHeight,
	unsigned* subImage, int subWidth, int subHeight,
	double reflection, double compression, double stretch,
	double accuracy, int iterations, int &found_x, int &found_y)
{
	const int pointsCount = 3;
	const int coordsCount = 2;
	const int bestIndex = 0;
	const int middleIndex = 1;
	const int worstIndex = 2;

	//��������� ��������� ��������� x � y
#pragma region
	int maxX = mainWidth - subWidth;
	int maxY = mainHeight - subHeight;

	int blockSize;
	int gridSize;
	if (iterations * pointsCount * coordsCount < maxThreads)
	{
		blockSize = iterations * pointsCount * coordsCount;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (iterations * pointsCount * coordsCount) / maxThreads + 1;
	}
	dim3 threads(blockSize);
	dim3 blocks(gridSize);

	hiprandState_t* states;
	hipMalloc((void**)&states, sizeof(hiprandState_t) * iterations * pointsCount * coordsCount);
	int* coords = new int[iterations * pointsCount * coordsCount];
	int* dev_coords;
	hipMalloc((void**)&dev_coords, sizeof(int) * iterations * pointsCount * coordsCount);

	getRandomNumbers_kernel <<<blocks, threads>>> (time(0), states, iterations, pointsCount, coordsCount, maxX, maxY, dev_coords);
	hipMemcpy(coords, dev_coords, sizeof(int) * iterations * pointsCount * coordsCount, hipMemcpyDeviceToHost);

#pragma endregion

	//���������� ��������� �� x � y
#pragma region
	if (iterations * pointsCount < maxThreads)
	{
		blockSize = iterations * pointsCount;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (iterations * pointsCount) / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	int* xCoords = new int[iterations * pointsCount];
	int* dev_xCoords;
	hipMalloc((void**)&dev_xCoords, sizeof(int) * iterations * pointsCount);
	int* yCoords = new int[iterations * pointsCount];
	int* dev_yCoords;
	hipMalloc((void**)&dev_yCoords, sizeof(int) * iterations * pointsCount);
	hipMalloc((void**)&dev_coords, sizeof(int) * iterations * pointsCount * coordsCount);
	hipMemcpy(dev_coords, coords, sizeof(int) * iterations * pointsCount * coordsCount, hipMemcpyHostToDevice);

	getCoords <<<threads, blocks>>> (iterations, pointsCount, coordsCount, dev_coords, dev_xCoords, dev_yCoords);
	hipMemcpy(xCoords, dev_xCoords, sizeof(int) * iterations * pointsCount, hipMemcpyDeviceToHost);
	hipMemcpy(yCoords, dev_yCoords, sizeof(int) * iterations * pointsCount, hipMemcpyDeviceToHost);

	hipFree(states);
	hipFree(dev_coords);
	hipFree(dev_xCoords);
	hipFree(dev_yCoords);
#pragma endregion

	//��������� �������� ������� ��� ���� �����
#pragma region
	if (subWidth * subHeight < maxThreads)
	{
		threads = dim3(subWidth * subHeight);
		blocks = dim3();
	}
	else
	{
		threads = dim3(maxThreads);
		blocks = dim3((subWidth * subHeight + maxThreads - 1) / maxThreads);
	}

	unsigned* dev_mainImage;
	hipMalloc((void**)&dev_mainImage, sizeof(unsigned) * mainWidth * mainHeight);
	hipMemcpy(dev_mainImage, mainImage, sizeof(unsigned) * mainWidth * mainHeight, hipMemcpyHostToDevice);

	unsigned* dev_subImage;
	hipMalloc((void**)&dev_subImage, sizeof(unsigned) * subWidth * subHeight);
	hipMemcpy(dev_subImage, subImage, sizeof(unsigned) * subWidth * subHeight, hipMemcpyHostToDevice);

	long double* functions = new long double[iterations * pointsCount];
	for (int i = 0; i < iterations * pointsCount; i++)
	{
		long double* numerators = new long double[subWidth * subHeight];
		long double* dev_numerators;
		hipMalloc((void**)&dev_numerators, sizeof(long double) * blocks.x);
		long double* subMultipliers = new long double[blocks.x];
		long double* dev_subMultipliers;
		hipMalloc((void**)&dev_subMultipliers, sizeof(long double) * blocks.x);
		long double* mainMultipliers = new long double[blocks.x];
		long double* dev_mainMultipliers;
		hipMalloc((void**)&dev_mainMultipliers, sizeof(long double) * blocks.x);

		int x = xCoords[i];
		int y = yCoords[i];

		getFunctionParameters <<<blocks, threads>>> (xCoords[i], yCoords[i], dev_mainImage,
			mainWidth, mainHeight, dev_subImage, subWidth, subHeight, dev_numerators, 
			dev_subMultipliers, dev_mainMultipliers);

		hipMemcpy(numerators, dev_numerators, sizeof(long double) * blocks.x, hipMemcpyDeviceToHost);
		hipMemcpy(subMultipliers, dev_subMultipliers, sizeof(long double) * blocks.x, hipMemcpyDeviceToHost);
		hipMemcpy(mainMultipliers, dev_mainMultipliers, sizeof(long double) * blocks.x, hipMemcpyDeviceToHost);

		long double numerator = 0;
		long double subMultiplier = 0;
		long double mainMultiplier = 0;
		for (int i = 0; i < blocks.x; i++)
		{
			numerator += numerators[i];
			subMultiplier += subMultipliers[i];
			mainMultiplier += mainMultipliers[i];
		}

		long double function = numerator / (sqrt(subMultiplier * mainMultiplier));
		functions[i] = function;

		hipFree(dev_numerators);
		hipFree(dev_subMultipliers);
		hipFree(dev_mainMultipliers);
	}
	hipFree(dev_mainImage);
	hipFree(dev_subImage);
#pragma endregion

	//���������� �������� ��� ���� ������� �����
	long double found_f = 0;
	int bestX, bestY, midX, midY, worstX, worstY;
	long double bestF, midF, worstF;
	for (int i = 0; i < iterations; i++)
	{
		int p_xCoords[pointsCount];
		int p_yCoords[pointsCount];
		long double funcs[pointsCount];
		for (int j = 0; j < pointsCount; j++)
		{
			p_xCoords[j] = xCoords[i * pointsCount + j];
			p_yCoords[j] = yCoords[i * pointsCount + j];
			funcs[j] = functions[i * pointsCount + j];
		}

		bool isFound = false;
		int count = 0;
		while (!isFound)
		{
			SortCoords(p_xCoords, p_yCoords, funcs, pointsCount);
			bestX = p_xCoords[bestIndex];
			bestY = p_yCoords[bestIndex];
			bestF = funcs[bestIndex];
			midX = p_xCoords[middleIndex];
			midY = p_yCoords[middleIndex];
			midF = funcs[middleIndex];
			worstX = p_xCoords[worstIndex];
			worstY = p_yCoords[worstIndex];
			worstF = funcs[worstIndex];



			break;//
		}
	}


	found_x = bestX;
	found_y = bestY;



	return 0;
}

void SortCoords(int* xCoords, int* yCoords, long double* functions, int pointsCount)
{
	for (int i = 0; i < pointsCount; i++)
	{
		for (int j = i + 1; j < pointsCount; j++)
		{
			if (functions[i] < functions[j])
			{
				long double temp_f = functions[i];
				int temp_x = xCoords[i];
				int temp_y = yCoords[i];

				functions[i] = functions[j];
				xCoords[i] = xCoords[j];
				yCoords[i] = yCoords[j];

				functions[j] = temp_f;
				xCoords[j] = temp_x;
				yCoords[j] = temp_y;
			}
		}
	}
}

