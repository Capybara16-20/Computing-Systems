﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#define N 10

__global__ void kernel()
{
	//обработать данные, находящиеся по этому индексу
	int tid = blockIdx.x;
	printf("Block number %d\n", tid);
}

int main(void)
{
	int* dev_a;
	// выделение памяти на GPU
	hipMalloc((void**)&dev_a, sizeof(int));
	kernel<<<N, 1>>>(); //вызов ядра
	hipFree(dev_a); // освобождение памяти на устройстве GPU
}
