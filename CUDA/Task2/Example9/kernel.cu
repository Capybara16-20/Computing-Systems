﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <locale.h>
#define BS 16//Размер блока
#define BC 4//Количество блоков

__global__ void kern(short* data)
{
    //Создается массив short в разделяемой памяти
    __shared__ short a[BS];
    int idx = blockIdx.x * BS + threadIdx.x;
    //Копируем из глобальной памяти в разделяемую
    a[threadIdx.x] = data[idx];
    //Перед использованием надо быть уверенным,
    //что все данные скопированы
    __syncthreads(); //Синхронизируем
    data[idx] = a[threadIdx.x] + a[(threadIdx.x + 1) % BS]; //Используем
}

int main()
{
    setlocale(LC_ALL, "Rus");
    short data[BC * BS];
    short* dev_data;
    hipMalloc((void**)&dev_data, BC * BS * sizeof(short));
    for (int i = 0; i < BC * BS; i++)
        data[i] = i;
    hipMemcpy(dev_data, data, BC * BS * sizeof(short), hipMemcpyHostToDevice);
    kern <<<BC, BS>>>(dev_data);
    hipMemcpy(data, dev_data, BC * BS * sizeof(short), hipMemcpyDeviceToHost);
    for (int i = 0; i < BC * BS; i++)
        printf("%hd\n", data[i]);
    hipFree(dev_data);
    return 0;
}


