#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <locale.h>
#include <windows.h>
#include <time.h>
#define SIZE (100 * 1024 * 1024)

void* big_random_block(int size)
{
	unsigned char* data = (unsigned char*)malloc(size);
	for (int i = 0; i < size; i++)
		data[i] = rand();
	return data;
}

__global__ void histo_kernel(unsigned char* buffer,
	long size, unsigned int* histo) 
{
	__shared__ unsigned int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < size) 
	{
		atomicAdd(&temp[buffer[i]], 1);
		i += stride;
	}
	__syncthreads();
	atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}

/*__global__ void histo_kernel(unsigned char* buffer,
	long size, unsigned int* histo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < size) 
	{
		atomicAdd(&histo[buffer[i]], 1);
		i += stride;
	}
}*/

int main(void) 
{
	setlocale(LC_ALL, "RUS");
	unsigned char* buffer = (unsigned char*)big_random_block(SIZE);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	unsigned char* dev_buffer;
	unsigned int* dev_histo;
	hipMalloc((void**)&dev_buffer, SIZE);
	hipMalloc((void**)&dev_histo, 256 * sizeof(int));
	hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);
	hipMemset(dev_histo, 0, 256 * sizeof(int));

	//наилучшая производительность достигается,
	//когда количество запускаемых блоков
	//в 2 раза больше количества мультипроцессоров
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int blocks = prop.multiProcessorCount;
	histo_kernel <<<blocks * 2, 256>>>(dev_buffer, SIZE, dev_histo);
	unsigned int histo[256];
	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Время выполнения: %3.1f миллисекунд\n", elapsedTime);
	long histoCount = 0;
	for (int i = 0; i < 256; i++)
		histoCount += histo[i];
	printf("Сумма гистограммы на GPU: %ld\n", histoCount);

	for (int i = 0; i < SIZE; i++)
			histo[buffer[i]]--;
	for (int i = 0; i < 256; i++)
		if (histo[i] != 0)
			printf("Ошибка в столбце %d\n", i);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_histo);
	hipFree(dev_buffer);
	free(buffer);
	return 0;
}


