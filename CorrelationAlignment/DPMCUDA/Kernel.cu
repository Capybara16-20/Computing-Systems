#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int maxThreads = 512;

__global__ void getGrayImage_kernel(unsigned* src, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x < width) && (y < height))
	{
		unsigned pixel = src[y * width + x];
		unsigned gray = ((pixel & 0x00FF0000 >> 0x10) +
			(pixel & 0x0000FF00 >> 0x8) + (pixel & 0x000000FF)) / 3;
		src[y * width + x] = 0xFF000000 | (gray << 0x10) | (gray << 0x8) | gray;
	}
}

__global__ void getRandomNumbers_kernel(unsigned int seed, hiprandState_t* states, 
	int iterations, int pointsCount, int coordsCount, int maxX, int maxY, int* coords)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < iterations * pointsCount * coordsCount)
	{
		hiprand_init(seed, i, 0, &states[i]);
		if (i % 2 == 0)
			coords[i] = hiprand(&states[i]) % maxX;
		else
			coords[i] = hiprand(&states[i]) % maxY;
	}
}

__global__ void getCoords(int iterations, int pointsCount, int coordsCount, int* coords, int* xCoords, int* yCoords)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = tid * coordsCount;
	xCoords[tid] = coords[i];
	yCoords[tid] = coords[i + 1];
}

__global__ void getFunctionParameters(int x, int y, unsigned* main, int mainWidth, 
	int mainHeight, unsigned* sub, int subWidth, int subHeight, 
	long double* numerators, long double* subMultipliers, long double* mainMultipliers)
{
	__shared__ long double temp_numerators[maxThreads];
	__shared__ long double temp_subMultipliers[maxThreads];
	__shared__ long double temp_mainMultipliers[maxThreads];
	int tid = threadIdx.x;
	int sub_index = blockIdx.x * blockDim.x + threadIdx.x;
	int y_offset = sub_index / subWidth;
	int x_offset = sub_index % subWidth;
	int main_i = mainWidth * (y + y_offset) + x + x_offset;
	long double temp_numerator = 0;
	long double temp_subMultiplier = 0;
	long double temp_mainMultiplier = 0;
	while (sub_index < subWidth * subHeight)
	{
		unsigned subPixel = sub[sub_index];
		long double subBrightness = 0.299 * ((subPixel & 0x00ff0000) >> 16)
			+ 0.587 * ((subPixel & 0x0000ff00) >> 8) + 0.114 * (subPixel & 0x000000ff);

		unsigned mainPixel = main[main_i];
		long double mainBrightness = 0.299 * ((mainPixel & 0x00ff0000) >> 16)
			+ 0.587 * ((mainPixel & 0x0000ff00) >> 8) + 0.114 * (mainPixel & 0x000000ff);

		temp_numerator += subBrightness * mainBrightness;
		temp_subMultiplier += subBrightness * subBrightness;
		temp_mainMultiplier += mainBrightness * mainBrightness;

		sub_index += blockDim.x * gridDim.x;
	}
	temp_numerators[tid] = temp_numerator;
	temp_subMultipliers[tid] = temp_subMultiplier;
	temp_mainMultipliers[tid] = temp_mainMultiplier;
	__syncthreads();
	for (int s = 1; s < blockDim.x; s *= 2)
	{
		if (tid % (2 * s) == 0)
		{
			temp_numerators[tid] += temp_numerators[tid + s];
			temp_subMultipliers[tid] += temp_subMultipliers[tid + s];
			temp_mainMultipliers[tid] += temp_mainMultipliers[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		numerators[blockIdx.x] = temp_numerators[0];
		subMultipliers[blockIdx.x] = temp_subMultipliers[0];
		mainMultipliers[blockIdx.x] = temp_mainMultipliers[0];
	}
}

int getGrayImage(unsigned* srcImage, int width, int height)
{
	unsigned* dev_srcImage;
	size_t size = sizeof(unsigned) * width * height;
	if (hipMalloc((void**)&dev_srcImage, size) != hipError_t::hipSuccess)
		return -1;
	if (hipMemcpy(dev_srcImage, srcImage, size,
		hipMemcpyHostToDevice) != hipError_t::hipSuccess)
		return -1;

	dim3 threads;
	dim3 blocks;
	if (width * height < maxThreads)
	{
		threads = dim3(width, height);
		blocks = dim3();
	}
	else
	{
		threads = dim3(maxThreads, maxThreads);
		blocks = dim3((width + maxThreads - 1) / maxThreads, (height + maxThreads - 1) / maxThreads);
	}

	getGrayImage_kernel <<<blocks, threads>>> (dev_srcImage, width, height);

	if (hipMemcpy(srcImage, dev_srcImage, size, 
		hipMemcpyDeviceToHost) != hipError_t::hipSuccess)
		return -1;

	hipFree(dev_srcImage);
	return 0;
}

int findImage(unsigned* mainImage, int mainWidth, int mainHeight,
	unsigned* subImage, int subWidth, int subHeight,
	double reflection, double compression, double stretch,
	double accuracy, int iterations, int &found_x, int &found_y)
{
	const int pointsCount = 3;
	const int coordsCount = 2;
	const int bestIndex = 0;
	const int middleIndex = 1;
	const int worstIndex = 2;

	//��������� ��������� ��������� x � y
#pragma region
	int maxX = mainWidth - subWidth;
	int maxY = mainHeight - subHeight;

	int blockSize;
	int gridSize;
	/*if (iterations * pointsCount * coordsCount < maxThreads)
	{
		blockSize = iterations * pointsCount * coordsCount;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (iterations * pointsCount * coordsCount) / maxThreads + 1;
	}
	dim3 threads(blockSize);
	dim3 blocks(gridSize);

	hiprandState_t* states;
	hipMalloc((void**)&states, sizeof(hiprandState_t) * iterations * pointsCount * coordsCount);
	int* coords = new int[iterations * pointsCount * coordsCount];
	int* dev_coords;
	hipMalloc((void**)&dev_coords, sizeof(int) * iterations * pointsCount * coordsCount);

	getRandomNumbers_kernel <<<blocks, threads>>> (time(0), states, iterations, pointsCount, coordsCount, maxX, maxY, dev_coords);
	hipMemcpy(coords, dev_coords, sizeof(int) * iterations * pointsCount * coordsCount, hipMemcpyDeviceToHost);

#pragma endregion

	//���������� ��������� �� x � y
#pragma region
	if (iterations * pointsCount < maxThreads)
	{
		blockSize = iterations * pointsCount;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (iterations * pointsCount) / maxThreads + 1;
	}
	threads = dim3(blockSize);
	blocks = dim3(gridSize);

	int* xCoords = new int[iterations * pointsCount];
	int* dev_xCoords;
	hipMalloc((void**)&dev_xCoords, sizeof(int) * iterations * pointsCount);
	int* yCoords = new int[iterations * pointsCount];
	int* dev_yCoords;
	hipMalloc((void**)&dev_yCoords, sizeof(int) * iterations * pointsCount);
	hipMalloc((void**)&dev_coords, sizeof(int) * iterations * pointsCount * coordsCount);
	hipMemcpy(dev_coords, coords, sizeof(int) * iterations * pointsCount * coordsCount, hipMemcpyHostToDevice);

	getCoords <<<threads, blocks>>> (iterations, pointsCount, coordsCount, dev_coords, dev_xCoords, dev_yCoords);
	hipMemcpy(xCoords, dev_xCoords, sizeof(int) * iterations * pointsCount, hipMemcpyDeviceToHost);
	hipMemcpy(yCoords, dev_yCoords, sizeof(int) * iterations * pointsCount, hipMemcpyDeviceToHost);

	hipFree(states);
	hipFree(dev_coords);
	hipFree(dev_xCoords);
	hipFree(dev_yCoords);*/

	int* xCoords = new int[iterations * pointsCount];
	int* yCoords = new int[iterations * pointsCount];
	for (int i = 0; i < iterations * pointsCount; i++)
	{
		xCoords[i] = rand() % maxX;
		yCoords[i] = rand() % maxY;
	}

#pragma endregion

	//��������� �������� ������� ��� ���� �����
#pragma region
	int blocksCount;
	int threadsCount;
	if (subWidth * subHeight < maxThreads)
	{
		blockSize = subWidth * subHeight;
		gridSize = 1;
	}
	else
	{
		blockSize = maxThreads;
		gridSize = (subWidth * subHeight + maxThreads - 1) / maxThreads;
	}

	unsigned* dev_mainImage;
	hipMalloc((void**)&dev_mainImage, sizeof(unsigned) * mainWidth * mainHeight);
	hipMemcpy(dev_mainImage, mainImage, sizeof(unsigned) * mainWidth * mainHeight, hipMemcpyHostToDevice);

	unsigned* dev_subImage;
	hipMalloc((void**)&dev_subImage, sizeof(unsigned) * subWidth * subHeight);
	hipMemcpy(dev_subImage, subImage, sizeof(unsigned) * subWidth * subHeight, hipMemcpyHostToDevice);

	long double* functions = new long double[iterations * pointsCount];
	for (int i = 0; i < iterations * pointsCount; i++)
	{
		functions[i] = GetFunctionValue(xCoords[i], yCoords[i], dev_mainImage, mainWidth,
			mainHeight, dev_subImage, subWidth, subHeight, blockSize, gridSize);
		//functions[i] = tempGetFunctionValue(xCoords[i], yCoords[i], mainImage, mainWidth, mainHeight, subImage, subWidth, subHeight);
	}
#pragma endregion

	//���������� �������� ��� ���� ������� �����
#pragma region
	long double found_f = 0;
	int best_x, best_y, middle_x, middle_y, worst_x, worst_y;
	long double best_f, middle_f, worst_f;
	for (int i = 0; i < iterations; i++)
	{
		int* p_xCoords = new int[pointsCount];
		int* p_yCoords = new int[pointsCount];
		long double funcs[pointsCount];
		for (int j = 0; j < pointsCount; j++)
		{
			p_xCoords[j] = xCoords[i * pointsCount + j];
			p_yCoords[j] = yCoords[i * pointsCount + j];
			funcs[j] = functions[i * pointsCount + j];
		}

		bool isFound = false;
		while (!isFound)
		{
			SortCoords(p_xCoords, p_yCoords, funcs, pointsCount);
			best_x = p_xCoords[bestIndex];
			best_y = p_yCoords[bestIndex];
			best_f = funcs[bestIndex];
			middle_x = p_xCoords[middleIndex];
			middle_y = p_yCoords[middleIndex];
			middle_f = funcs[middleIndex];
			worst_x = p_xCoords[worstIndex];
			worst_y = p_yCoords[worstIndex];
			worst_f = funcs[worstIndex];

			//������ ������ �������
			int gravityCenter_x = (best_x + middle_x) / (pointsCount - 1);
			int gravityCenter_y = (best_y + middle_y) / (pointsCount - 1);
			long double gravityCenter_f = GetFunctionValue(gravityCenter_x, gravityCenter_y, dev_mainImage, mainWidth,
				mainHeight, dev_subImage, subWidth, subHeight, blockSize, gridSize);

			//������� ���������
			bool isImageFound = IsImageFound(funcs, pointsCount, gravityCenter_f, accuracy);
			if (isImageFound)
			{
				isFound = true;

				if (funcs[bestIndex] > found_f)
				{
					found_x = p_xCoords[bestIndex];
					found_y = p_yCoords[bestIndex];
					found_f = funcs[bestIndex];
				}

				break;
			}

			if (abs(gravityCenter_x - worst_x) <= 1 && abs(gravityCenter_y - worst_y) <= 1)
				break;

			//"���������"
			int reflected_x = GetReflectedCoord(gravityCenter_x, worst_x, reflection, maxX);
			int reflected_y = GetReflectedCoord(gravityCenter_y, worst_y, reflection, maxY);

			long double reflected_f = GetFunctionValue(reflected_x, reflected_y, dev_mainImage, mainWidth,
				mainHeight, dev_subImage, subWidth, subHeight, blockSize, gridSize);

			if (reflected_f >= best_f) //"����������"
			{
				int stretched_x = GetStretchedCoord(gravityCenter_x, reflected_x, stretch, maxX);
				int stretched_y = GetStretchedCoord(gravityCenter_y, reflected_y, stretch, maxY);

				long double stretched_f = GetFunctionValue(stretched_x, stretched_y, dev_mainImage, mainWidth,
					mainHeight, dev_subImage, subWidth, subHeight, blockSize, gridSize);

				if (stretched_f > best_f)
				{
					p_xCoords[worstIndex] = stretched_x;
					p_yCoords[worstIndex] = stretched_y;
					funcs[worstIndex] = stretched_f;
				}
				else
				{
					p_xCoords[worstIndex] = reflected_x;
					p_yCoords[worstIndex] = reflected_y;
					funcs[worstIndex] = reflected_f;
				}
			}
			else if (reflected_f >= worst_f && reflected_f < middle_f) //"������"
			{
				int compressed_x = GetCompressedCoord(gravityCenter_x, worst_x, compression, maxX);
				int compressed_y = GetCompressedCoord(gravityCenter_y, worst_y, compression, maxY);
				
				long double compressed_f = GetFunctionValue(compressed_x, compressed_y, dev_mainImage, mainWidth,
					mainHeight, dev_subImage, subWidth, subHeight, blockSize, gridSize);

				p_xCoords[worstIndex] = compressed_x;
				p_yCoords[worstIndex] = compressed_y;
				funcs[worstIndex] = compressed_f;
			}
			else if (reflected_f >= middle_f && reflected_f < best_f) //������ ��������� �������
			{
				p_xCoords[worstIndex] = reflected_x;
				p_yCoords[worstIndex] = reflected_y;
				funcs[worstIndex] = reflected_f;
			}
			else //"��������"
			{
				p_xCoords = GetReducedCoords(p_xCoords, best_x, pointsCount, maxX);
				p_yCoords = GetReducedCoords(p_yCoords, best_y, pointsCount, maxY);

				for (int i = 0; i < pointsCount; i++)
					funcs[i] = GetFunctionValue(p_xCoords[i], p_yCoords[i], dev_mainImage, mainWidth,
						mainHeight, dev_subImage, subWidth, subHeight, blockSize, gridSize);
			}

		}
	}
	hipFree(dev_mainImage);
	hipFree(dev_subImage);

	int a = found_x;
	int b = found_y;
	long double f = found_f;
#pragma endregion
	return 0;
}

void SortCoords(int* xCoords, int* yCoords, long double* functions, int pointsCount)
{
	for (int i = 0; i < pointsCount; i++)
	{
		for (int j = i + 1; j < pointsCount; j++)
		{
			if (functions[i] < functions[j])
			{
				long double temp_f = functions[i];
				int temp_x = xCoords[i];
				int temp_y = yCoords[i];

				functions[i] = functions[j];
				xCoords[i] = xCoords[j];
				yCoords[i] = yCoords[j];

				functions[j] = temp_f;
				xCoords[j] = temp_x;
				yCoords[j] = temp_y;
			}
		}
	}
}

long double GetFunctionValue(int x, int y, unsigned* dev_mainImage, int mainWidth, int mainHeight,
	unsigned* dev_subImage, int subWidth, int subHeight, int blockSize, int gridSize)
{
	dim3 threads(blockSize);
	dim3 blocks(gridSize);

	long double* numerators = new long double[blocks.x];
	long double* dev_numerators;
	hipMalloc((void**)&dev_numerators, sizeof(long double) * blocks.x);
	long double* subMultipliers = new long double[blocks.x];
	long double* dev_subMultipliers;
	hipMalloc((void**)&dev_subMultipliers, sizeof(long double) * blocks.x);
	long double* mainMultipliers = new long double[blocks.x];
	long double* dev_mainMultipliers;
	hipMalloc((void**)&dev_mainMultipliers, sizeof(long double) * blocks.x);

	getFunctionParameters <<<blocks, threads>>> (x, y, dev_mainImage,
		mainWidth, mainHeight, dev_subImage, subWidth, subHeight, dev_numerators,
		dev_subMultipliers, dev_mainMultipliers);

	hipMemcpy(numerators, dev_numerators, sizeof(long double) * blocks.x, hipMemcpyDeviceToHost);
	hipMemcpy(subMultipliers, dev_subMultipliers, sizeof(long double) * blocks.x, hipMemcpyDeviceToHost);
	hipMemcpy(mainMultipliers, dev_mainMultipliers, sizeof(long double) * blocks.x, hipMemcpyDeviceToHost);

	hipFree(dev_numerators);
	hipFree(dev_subMultipliers);
	hipFree(dev_mainMultipliers);

	long double numerator = 0;
	long double subMultiplier = 0;
	long double mainMultiplier = 0;
	for (int i = 0; i < blocks.x; i++)
	{
		numerator += numerators[i];
		subMultiplier += subMultipliers[i];
		mainMultiplier += mainMultipliers[i];
	}

	long double function = numerator / (sqrt(subMultiplier * mainMultiplier));
	return function;
}

bool IsImageFound(long double* functions, int pointsCount,
	long double gravityCenter_f, double accuracy)
{
	long double currentAccuracy = 0;
	for (int i = 0; i < pointsCount; i++)
		currentAccuracy += pow(functions[i] - gravityCenter_f, 2);
	currentAccuracy = sqrt(currentAccuracy / pointsCount);

	return currentAccuracy <= accuracy;
}

int GetReflectedCoord(int gravityCenter, int worst, double reflection, int max)
{
	int reflected = gravityCenter + (gravityCenter - worst) * reflection;

	if (reflected > max)
		reflected = max;
	if (reflected < 0)
		reflected = 0;

	return reflected;
}

int GetStretchedCoord(int gravityCenter, int reflected, double stretch, int max)
{
	int stretched = gravityCenter + (reflected - gravityCenter) * stretch;

	if (stretched > max)
		stretched = max;
	if (stretched < 0)
		stretched = 0;

	return reflected;
}

int GetCompressedCoord(int gravityCenter, int worst, double compression, int max)
{
	int compressed = gravityCenter + (worst - gravityCenter) * compression;

	if (compressed > max)
		compressed = max;
	if (compressed < 0)
		compressed = 0;

	return compressed;
}

int* GetReducedCoords(int* coords, int best, int pointsCount, int max)
{
	int* reducedCoords = new int[pointsCount];
	for (int i = 0; i < pointsCount; i++)
	{
		int reducedCoord = best + (coords[i] - best) * 0.5;

		if (reducedCoord > max)
			reducedCoord = max;
		if (reducedCoord < 0)
			reducedCoord = 0;

		reducedCoords[i] = reducedCoord;
	}

	return reducedCoords;
}
