#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <locale.h>
#include <windows.h>
#include <time.h>
#define SIZE (100 * 1024 * 1024)

void* big_random_block(int size)
{
	unsigned char* data = (unsigned char*)malloc(size);
	for (int i = 0; i < size; i++)
		data[i] = rand();
	return data;
}

int main(void) 
{
	setlocale(LC_ALL, "RUS");
	unsigned char* buffer = (unsigned char*)big_random_block(SIZE);
	clock_t start, stop; //объявление переменных для замера времени
	start = clock(); //начало замера
	unsigned int histo[256];
	for (int i = 0; i < 256; i++) 
		histo[i] = 0;
	for (int i = 0; i < SIZE; i++) 
		histo[buffer[i]]++;
	stop = clock(); //окончание замера
	float elapsedTime = (float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f;
	printf("Время выполнения: %3.1f миллисекунд\n", elapsedTime);
	long histoCount = 0;
	for (int i = 0; i < 256; i++) 
		histoCount += histo[i];
	printf("Сумма гистограммы: %ld\n", histoCount);
	free(buffer); //освобождение памяти
	return 0;
}
