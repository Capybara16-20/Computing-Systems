﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#define N 10

__global__ void add(int* a, int* b, int* c)
{
	int tid = threadIdx.x; //обработать данные, находящиеся по этому индексу
	if (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		printf("Thread number %d write %d in array on %d position\n", tid, c[tid], tid);
	}
}

int main()
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;
	// выделение памяти на GPU
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	// заполнение массивов ‘a’ и ‘b’ CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}
	// копирование массивов 'a' и'b' на GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	add<<<1, N>>>(dev_a, dev_b, dev_c); //вызов ядра
	// копирование массива 'c' с GPU на CPU
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	// отображение элементов результирующего массива
	printf("\n");
	for (int i = 0; i < N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	hipFree(dev_a); // освобождение памяти на GPU
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}


