﻿#include <hip/device_functions.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include<locale.h>
#define N 10
#define K 3

__global__ void mult(int* matrix, int* result)
{
	int tid_block = blockIdx.x;
	int tid_thread = threadIdx.x;
	if (tid_block < N && tid_thread < N)
		result[tid_block * N + tid_thread] = matrix[tid_block * N + tid_thread] * 3;
}

int main(void)
{
	setlocale(LC_ALL, "Rus");
	int matrix[N * N];
	int* dev_matrix;
	int result[N * N];
	int* dev_result;
	hipMalloc((void**)&dev_matrix, N * N * sizeof(int));
	hipMalloc((void**)&dev_result, N * N * sizeof(int));
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			matrix[i * N + j] = i + j;
	hipMemcpy(dev_matrix, matrix, N * N * sizeof(int), hipMemcpyHostToDevice);
	printf("Матрица:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%d\t", matrix[i * N + j]);
		printf("\n");
	}
	mult<<<N, N>>>(dev_matrix, dev_result);
	hipMemcpy(result, dev_result, N * N * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nРезультат:\n");
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%d\t", result[i * N + j]);
		printf("\n");
	}
	hipFree(dev_matrix);
	hipFree(dev_result);
	return 0;
}

