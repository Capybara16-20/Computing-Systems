﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#define N 10
#define M 3

__global__ void kernel()
{
	// обработать данные по этому индексу
	int tid_block = blockIdx.x;
	int tid_thread = threadIdx.x;
	printf("Block number %d. Thread number %d\n", tid_block, tid_thread);
}

int main(void)
{
	int* dev_a;
	// выделение памяти на GPU
	hipMalloc((void**)&dev_a, sizeof(int));
	kernel <<<M, N>>>(); //вызов ядра
	// освобождение памяти на устройстве GPU
	hipFree(dev_a);
	return 0;
}