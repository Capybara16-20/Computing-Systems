﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <locale.h>
#include <math.h>
#define N 32

__global__ void pow(int k, unsigned long long* c)
{
	int tid = threadIdx.x;
	c[tid] = pow(k, tid);
}

int main()
{
	setlocale(LC_ALL, "Rus");

	int k;
	printf("Введите номер варианта: ");
	scanf("%d", &k);
	k += 3;

	printf("k = %d\n\n", k);

	unsigned long long res[N];
	unsigned long long* dev_res;
	hipMalloc((void**)&dev_res, N * sizeof(unsigned long long));
	hipMemcpy(dev_res, res, N * sizeof(unsigned long long), hipMemcpyHostToDevice);

	pow<<<1, N>>>(k, dev_res);

	hipMemcpy(res, dev_res, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);

	printf("Степени числа %d\n", k);
	for (int i = 0; i < N; i++)
	{
		printf("%d ^ %d ", k, i);
		if (res[i] == ULLONG_MAX)
			printf(" > %zu\n", res[i]);
		else
			printf(" = %zu\n", res[i]);
	}

	hipFree(dev_res);
	return 0;
}


